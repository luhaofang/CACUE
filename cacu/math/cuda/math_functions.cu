#include "hip/hip_runtime.h"
/*
 Copyright (c) 2016, David lu
 All rights reserved.

 Redistribution and use in source and binary forms, with or without
 modification, are permitted provided that the following conditions are met:
 * Redistributions of source code must retain the above copyright
 notice, this list of conditions and the following disclaimer.
 * Redistributions in binary form must reproduce the above copyright
 notice, this list of conditions and the following disclaimer in the
 documentation and/or other materials provided with the distribution.
 * Neither the name of the <organization> nor the
 names of its contributors may be used to endorse or promote products
 derived from this software without specific prior written permission.

 THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY
 EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY
 DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "../../config.h"

#ifdef __PARALLELTYPE__
#if __PARALLELTYPE__ == __CUDA__

#include "../../tensor/cuda/cuda_log.h"

namespace cacu {

__global__ void _k_CACU_SAXPY_ATOMIC_CUDA(float *x, float a, float *y,
		const int length) {

	int tid = threadIdx.x;
	int bid = blockIdx.x;

	int threadid = bid * THREADNUM + tid;

	for (int i = threadid; i < length; i += BLOCKNUM * THREADNUM) {

		atomicAdd(y + i, a * x[i]);
	}

}

extern "C" void cacu_saxpy_atomic_cuda(float *x, float a, float *y,
		const int length) {
	_k_CACU_SAXPY_ATOMIC_CUDA<<<BLOCKNUM, THREADNUM, 0>>>(x, a, y, length);

	CUDA_CHECK(hipDeviceSynchronize());
}

__global__ void _k_CACU_ISAXB_CUDA(float *x, const int channel, const int width, const int height, const float a,
		int *index_, const float b, float *y) {

	int tid = threadIdx.x;
	int bid = blockIdx.x;

	int threadid = bid * THREADNUM + tid;
	float *xp,*yp;
	int c_length = width * height;
	int length = channel* c_length;

	if (index_[0] >= 0) {

		for (int i = threadid; i < length; i += BLOCKNUM * THREADNUM) {
			y[i] = x[i];
		}

		__syncthreads();

		for (int i = threadid ; i < c_length; i += BLOCKNUM * THREADNUM)
		{
			xp = x + i;
			yp = y + i;
			if (tid == 0)
				yp[index_[i] * c_length] = a * xp[index_[i] * c_length] + b;
		}
	}
	else {
		for (int i = threadid; i < length; i += BLOCKNUM * THREADNUM) {
			y[i] = 0;
		}
	}
}

/**
 * @cacu_isaxdb_cuda
 * y[index] = x[index]*a + b
 */
extern "C" void cacu_isaxb_cuda(float *x, const int channel, const int width, const int height, const float a,
		int *index_, const float b, float *y) {

	_k_CACU_ISAXB_CUDA<<<BLOCKNUM, THREADNUM, 0>>>(x, channel, width, height, a, index_, b, y);

	CUDA_CHECK(hipDeviceSynchronize());

}

__global__ void _k_ARGMAX_CUDA(float *x, const int length,
		unsigned int *index_) {

	__shared__ float shared_data[THREADNUM];

	__shared__ unsigned int index_data[THREADNUM];

	int tid = threadIdx.x;

	int max_length = THREADNUM;
	if (THREADNUM > length)
		max_length = length;

	if (tid < max_length) {
		shared_data[tid] = x[tid];
		index_data[tid] = tid;
	}

	for (unsigned int i = tid; i < length; i += THREADNUM) {
		if (x[i] > shared_data[tid]) {
			shared_data[tid] = x[i];
			index_data[tid] = i;
		}
	}

	__syncthreads();

	if (tid == 0) {

		for (int i = 1; i < max_length; ++i) {
			if (shared_data[0] < shared_data[i]) {
				shared_data[0] = shared_data[i];
				index_data[0] = index_data[i];
			}
		}
		index_[0] = index_data[0];
	}
}

extern "C" void cacu_argmax_cuda(float *x, const int length,
		unsigned int *index_) {
	_k_ARGMAX_CUDA<<<1, THREADNUM, 0>>>(x, length, index_);

	CUDA_CHECK(hipDeviceSynchronize());
}

__global__ void _k_CACU_TRANSPOSE_CUDA(float *mtx, const int m, const int n,
		const int length) {


}


extern "C" void cacu_transpose_cuda(float *mtx, const int m, const int n, const int length) {

}

__global__ void _k_CACU_CLIP_VEC_CUDA(float *data, const float threshold,
		const int length) {

	int tid = threadIdx.x;
	int bid = blockIdx.x;

	int threadid = bid * THREADNUM + tid;

	for (int i = threadid; i < length; i += BLOCKNUM * THREADNUM) {

		data[i] = data[i] * (abs(data[i]) >= threshold);
	}
}

extern "C" void cacu_clip_vec_cuda(float *data, const float threshold,
		const int length) {
	_k_CACU_CLIP_VEC_CUDA<<<BLOCKNUM, THREADNUM, 0>>>(data, threshold, length);

	CUDA_CHECK(hipDeviceSynchronize());
}

__global__ void _k_CACU_ABS_CUDA(float *x, const int length, float *y) {

	int tid = threadIdx.x;
	int bid = blockIdx.x;

	int threadid = bid * THREADNUM + tid;

	for (int i = threadid; i < length; i += BLOCKNUM * THREADNUM) {

		y[i] = abs(x[i]);
	}
}

extern "C" void cacu_abs_cuda(float *x, const int length, float *y)
{
	_k_CACU_ABS_CUDA<<<BLOCKNUM, THREADNUM, 0>>>(x, length, y);

	CUDA_CHECK(hipDeviceSynchronize());
}

__global__ void _k_CACU_ABS_GRAD_CUDA(float *x, float *diff, const int length) {

	int tid = threadIdx.x;
	int bid = blockIdx.x;

	int threadid = bid * THREADNUM + tid;

	for (int i = threadid; i < length; i += BLOCKNUM * THREADNUM) {

		diff[i] = (x[i] > 0) ? 1 : -1;
	}
}

extern "C" void cacu_abs_grad_cuda(float *x, float *diff, const int length)
{
	_k_CACU_ABS_GRAD_CUDA<<<BLOCKNUM, THREADNUM, 0>>>(x, diff, length);

	CUDA_CHECK(hipDeviceSynchronize());
}

}

#endif
#endif
