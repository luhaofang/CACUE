#include "hip/hip_runtime.h"
/*
 Copyright (c) 2016, David lu
 All rights reserved.

 Redistribution and use in source and binary forms, with or without
 modification, are permitted provided that the following conditions are met:
 * Redistributions of source code must retain the above copyright
 notice, this list of conditions and the following disclaimer.
 * Redistributions in binary form must reproduce the above copyright
 notice, this list of conditions and the following disclaimer in the
 documentation and/or other materials provided with the distribution.
 * Neither the name of the <organization> nor the
 names of its contributors may be used to endorse or promote products
 derived from this software without specific prior written permission.

 THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY
 EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY
 DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "../../config.h"

#ifdef __PARALLELTYPE__
#if __PARALLELTYPE__ == __CUDA__

#include "../../definition.h"
#include "../math_definition.h"
#include "../../tensor/cuda/cuda_log.h"

namespace cacu {

__global__ void _k_CACU_SUMBYSIZE_BYWIDTH_CUDA(const float_t *x, int heigth,
		int width, const float_t alpha, float_t *y, const float_t beta) {

	int tid = threadIdx.x;
	int bid = blockIdx.x;

	extern __shared__ float_t shared_data[];

	for (int i = bid; i < heigth; i += BLOCKNUM) {
		shared_data[tid] = 0;
		for (int j = tid; j < width; j += THREADNUM) {
			shared_data[tid] += x[i * width + j];
		}
		__syncthreads();

		int acc_length = THREADNUM / 2;
		while (acc_length > 0) {
			if (tid < acc_length)
				shared_data[tid] += shared_data[tid + acc_length];
			acc_length /= 2;
			__syncthreads();
		}

		if (tid == 0)
			y[i] = alpha * shared_data[0] + beta * y[i];
	}
}

__global__ void _k_CACU_SUMBYSIZE_BYHEIGHT_CUDA(const float_t *x, int height,
		int width, const float_t alpha, float_t *y, const float_t beta) {

	int tid = threadIdx.x;
	int bid = blockIdx.x;

	extern __shared__ float_t shared_data[];

	for (int i = bid; i < width; i += BLOCKNUM) {
		shared_data[tid] = 0;
		for (int j = tid; j < height; j += THREADNUM)
		{
			shared_data[tid] += x[j * width + i];
		}
		__syncthreads();

		int acc_length = THREADNUM / 2;
		while (acc_length > 0) {

			if (tid < acc_length)
				shared_data[tid] += shared_data[tid + acc_length];
			acc_length /= 2;
			__syncthreads();
		}

		if (tid == 0)
			y[i] = alpha * shared_data[0] + beta * y[i];
	}
}

/**
 * (need FULLY tested!)
 * @cacu_sumbysize_cuda
 * sum by size:
 * accumulate the value by width or height , width is the matrix array's width dim which stored in row -major format.
 * sum by width y is (length/ width) height dim, sum by height y is width dim.
 */
extern "C" void cacu_sumbysize_cuda(SUM SUMTYPE, const float_t *x, int length,
		const float_t alpha, float_t *y, const float_t beta, int width) {

	int height = length / width;

	if (BYWIDTH == SUMTYPE)
		_k_CACU_SUMBYSIZE_BYWIDTH_CUDA<<<BLOCKNUM, THREADNUM,
		THREADNUM * sizeof(float_t)>>>(x, height, width, alpha, y, beta);
	else if (BYHEIGHT == SUMTYPE)
		_k_CACU_SUMBYSIZE_BYHEIGHT_CUDA<<<BLOCKNUM, THREADNUM,
		THREADNUM * sizeof(float_t)>>>(x, height, width, alpha, y, beta);
	CUDA_CHECK(hipDeviceSynchronize());
}

__global__ void _k_CACU_CXSIZE_CUDA(const float_t *x, int length,
		const float_t *a, int size, float_t *y) {

	int tid = threadIdx.x;
	int bid = blockIdx.x;

	int threadid = bid * THREADNUM + tid;

	int block_size = length / size;

	for (int i = threadid; i < length; i += BLOCKNUM * THREADNUM) {
		y[i] = x[i] * a[i / block_size];
	}
}

/**
 * @cacu_cxsize_cuda
 * math y[i] = a[j]*x[i] :
 * x: length dim array list
 * a: size dim array list
 * a[j] is the corresponding scalar, j = i / (length / size).
 */
extern "C" void cacu_cxsize_cuda(const float_t *x, int length, const float_t *a,
		int size, float_t *y) {
	_k_CACU_CXSIZE_CUDA<<<BLOCKNUM, THREADNUM, 0>>>(x, length, a, size, y);
	CUDA_CHECK(hipDeviceSynchronize());
}

__global__ void _k_CACU_CDXSIZE_CUDA(const float_t *x, int length,
		const float_t *a, int size, float_t *y) {

	int tid = threadIdx.x;
	int bid = blockIdx.x;

	int threadid = bid * THREADNUM + tid;

	int block_size = length / size;

	for (int i = threadid; i < length; i += BLOCKNUM * THREADNUM) {
		y[i] = x[i] / a[i / block_size];
	}
}

/**
 * @cacu_cdxsize_cuda
 * math y[i] = x[i] / a[j] :
 * x: length dim array list
 * a: size dim array list
 * a[j] is the corresponding denominator, j = i / (length / size).
 */
extern "C" void cacu_cdxsize_cuda(const float_t *x, int length,
		const float_t *a, int size, float_t *y) {
	_k_CACU_CDXSIZE_CUDA<<<BLOCKNUM, THREADNUM, 0>>>(x, length, a, size, y);
	CUDA_CHECK(hipDeviceSynchronize());
}

__global__ void _k_CACU_SDXSIZE_CUDA(float_t *x, int length,
		const float_t a, const float_t b, float_t *y) {

	int tid = threadIdx.x;
	int bid = blockIdx.x;

	int threadid = bid * THREADNUM + tid;

	for (int i = threadid; i < length; i += BLOCKNUM * THREADNUM) {
		y[i] = (x[i] + a) / b;
	}
}

/**
 * @cacu_sdxsize
 * math y[i] = (x[i] + a) / b:
 * x is a length dim array list, a is the corresponding denominator.
 */
extern "C" void cacu_sdxsize_cuda(float_t *x, const int length,
		const float_t a, const float_t b, float_t *y) {
	_k_CACU_SDXSIZE_CUDA<<<BLOCKNUM, THREADNUM, 0>>>(x, length, a, b, y);
	CUDA_CHECK(hipDeviceSynchronize());
}

__global__ void _k_CACU_SSXPY_CUDA(const float_t *x, const float_t a, int size,
		const float_t *y, const float_t b, int length, float_t *z) {

	int tid = threadIdx.x;
	int bid = blockIdx.x;

	int threadid = bid * THREADNUM + tid;

	int block_size = length / size;

	for (int i = threadid; i < length; i += BLOCKNUM * THREADNUM) {
		z[i] = b * y[i] + a * x[i / block_size];
	}
}

/**
 * @cacu_ssxpy_cuda
 * math z[i] = a * x[j] + b * y[i] :
 * y: length dim array list
 * x: size dim array list
 * x[j] is the corresponding scalar, j = i / (length / size).
 * a & b are corresponding scalars for x, y
 */
extern "C" void cacu_ssxpy_cuda(const float_t *x, const float_t a, int size,
		const float_t *y, const float_t b, int length, float_t *z) {
	_k_CACU_SSXPY_CUDA<<<BLOCKNUM, THREADNUM, 0>>>(x, a, size, y, b, length, z);
	CUDA_CHECK(hipDeviceSynchronize());
}

__global__ void _k_CACU_SQR_CUDA(const float_t *x, int length, float_t *y) {

	int tid = threadIdx.x;
	int bid = blockIdx.x;

	int threadid = bid * THREADNUM + tid;

	for (int i = threadid; i < length; i += BLOCKNUM * THREADNUM) {
		y[i] = x[i] * x[i];
	}
}

/**
 * @cacu_sqr_cuda
 * math y[i] = x[i]^2 :
 */
extern "C" void cacu_sqr_cuda(const float_t *x, int length, float_t *y) {
	_k_CACU_SQR_CUDA<<<BLOCKNUM, THREADNUM, 0>>>(x, length, y);
	CUDA_CHECK(hipDeviceSynchronize());
}

__global__ void _k_CACU_ROOT_CUDA(const float_t *x, int length, float_t *y) {

	int tid = threadIdx.x;
	int bid = blockIdx.x;

	int threadid = bid * THREADNUM + tid;

	for (int i = threadid; i < length; i += BLOCKNUM * THREADNUM) {
		y[i] = sqrt(x[i]);
	}
}

/**
 * @cacu_root_cuda
 * math y[i] = sqrt(x[i]) :
 */
extern "C" void cacu_root_cuda(const float_t *x, int length, float_t *y) {
	_k_CACU_ROOT_CUDA<<<BLOCKNUM, THREADNUM, 0>>>(x, length, y);
	CUDA_CHECK(hipDeviceSynchronize());
}

__global__ void _k_CACU_STDBYCHANNEL_CUDA(const float_t *varience, int length,
		float_t *std, const float_t epsilon) {

	int tid = threadIdx.x;
	int bid = blockIdx.x;

	int threadid = bid * THREADNUM + tid;

	for (int i = threadid; i < length; i += BLOCKNUM * THREADNUM) {
		std[i] = sqrt(varience[i] + epsilon);
	}
}

/**
 * @cacu_stdbychannel_cuda
 * math std[i] = sqrt(varience[i] + epsilon) :
 */
extern "C" void cacu_stdbychannel_cuda(const float_t *varience, int length,
		float_t *std, const float_t epsilon) {
	_k_CACU_STDBYCHANNEL_CUDA<<<BLOCKNUM, THREADNUM, 0>>>(varience, length, std,
			epsilon);
	CUDA_CHECK(hipDeviceSynchronize());
}

__global__ void _k_CACU_BN_ROU_GRAD_CUDA(const float_t *x, const float_t *d_x,
		const float_t *mean, const float_t *std, int num, int length,
		int channel, float_t *d_rou) {

	int tid = threadIdx.x;
	int bid = blockIdx.x;

	extern __shared__ float_t shared_data[];

	int data_row, data_col;

	int cin_length = length / channel;

	int set;

	for (int i = bid; i < channel; i += BLOCKNUM)
	{
		shared_data[tid] = 0;
		for (int j = tid; j < cin_length * num; j += THREADNUM)
		{
			data_row = j / cin_length;
			data_col = j % cin_length;
			set = data_row * length + data_col + i * cin_length;
			shared_data[tid] += ((x[set] - mean[i]) * d_x[set]
					* (float_t(-0.5) / (std[i] * std[i] * std[i])));
		}

		__syncthreads();

		int acc_length = THREADNUM / 2;
		while (acc_length > 0) {
			if (tid < acc_length)
				shared_data[tid] += shared_data[tid + acc_length];
			acc_length /= 2;
			__syncthreads();
		}

		if (tid == 0)
			d_rou[i] += shared_data[0];
	}
}

/**
 * @cacu_bn_rou_grad_cuda
 * calculate the gradient of bn layer's rou
 * x: input feature
 * d_x: gradient of ^x
 * mean: mean of batch
 * std: standard deviation of batch
 * length: size of a feature map
 * d_rou: gradient of batch's variance
 */
extern "C" void cacu_bn_rou_grad_cuda(const float_t *x, const float_t *d_x,
		const float_t *mean, const float_t *std, int num, int length,
		int channel, float_t *d_rou) {

	_k_CACU_BN_ROU_GRAD_CUDA<<<BLOCKNUM, THREADNUM,
	THREADNUM * sizeof(float_t)>>>(x, d_x, mean, std, num, length, channel,
			d_rou);
	CUDA_CHECK(hipDeviceSynchronize());
}

__global__ void _k_CACU_BN_MU_GRAD_CUDA(const float_t *x, const float_t *d_x,
		const float_t *mean, const float_t *std, const float_t *d_rou, int num,
		int length, int channel, float_t *d_mean) {

	int tid = threadIdx.x;
	int bid = blockIdx.x;

	extern __shared__ float_t shared_data[];

	int data_row, data_col;

	int cin_length = length / channel;

	int set;

	int m = cin_length * num;

	for (int i = bid; i < channel; i += BLOCKNUM)
	{
		shared_data[tid] = 0;
		for (int j = tid; j < cin_length * num; j += THREADNUM)
		{
			data_row = j / cin_length;
			data_col = j % cin_length;
			set = data_row * length + data_col + i * cin_length;
			shared_data[tid] += ((d_x[set] / (-std[i]))
					+ ((d_rou[i] / m) * (float_t(-2) * (x[set] - mean[i]))));
		}

		__syncthreads();

		int acc_length = THREADNUM / 2;
		while (acc_length > 0) {
			if (tid < acc_length)
				shared_data[tid] += shared_data[tid + acc_length];
			acc_length /= 2;
			__syncthreads();
		}

		if (tid == 0)
			d_mean[i] += shared_data[0];
	}
}

/**
 * @cacu_bn_mu_grad
 * calculate the gradient of bn layer's mu
 * x: input feature
 * d_x: gradient of ^x
 * mean: mean of batch
 * std: standard deviation of batch
 * d_rou: gradient of batch's variance
 * length: size of a feature map
 * d_mean: gradient of batch's mean
 */
extern "C" void cacu_bn_mu_grad_cuda(const float_t *x, const float_t *d_x,
		const float_t *mean, const float_t *std, const float_t *d_rou, int num,
		int length, int channel, float_t *d_mean) {
	_k_CACU_BN_MU_GRAD_CUDA<<<BLOCKNUM, THREADNUM,
	THREADNUM * sizeof(float_t)>>>(x, d_x, mean, std, d_rou, num, length,
			channel, d_mean);
	CUDA_CHECK(hipDeviceSynchronize());
}

__global__ void _k_CACU_BN_DX_GRAD_CUDA(const float_t *x, const float_t *d_x,
		const float_t *mean, const float_t *std, const float_t *d_rou,
		const float_t *d_mean, int num, int length, int channel, float_t *dx) {

	int tid = threadIdx.x;
	int bid = blockIdx.x;

	int threadid = bid * THREADNUM + tid;

	int c;

	int cin_length = length / channel;

	int m = cin_length * num;

	for (int i = threadid; i < num * length; i += BLOCKNUM * THREADNUM) {

		c = (i % length) / cin_length;
		dx[i] = ((d_x[i] / std[c])
				+ d_rou[c] * (float_t(2) * (x[i] - mean[c]) / m)
				+ (d_mean[c] / m));
	}
}

/**
 * @cacu_bn_dx_grad_cuda
 * calculate the gradient of bn layer's dx
 * x: input feature
 * d_x: gradient of ^x
 * mean: mean of batch
 * std: standard deviation of batch
 * d_rou: gradient of batch's variance
 * d_mean: gradient of batch's mean
 * length: size of a feature map
 * dx: gradient of x
 */
extern "C" void cacu_bn_dx_grad_cuda(const float_t *x, const float_t *d_x,
		const float_t *mean, const float_t *std, const float_t *d_rou,
		const float_t *d_mean, int num, int length, int channel, float_t *dx) {
	_k_CACU_BN_DX_GRAD_CUDA<<<BLOCKNUM, THREADNUM, 0>>>(x, d_x, mean, std,
			d_rou, d_mean, num, length, channel, dx);
	CUDA_CHECK(hipDeviceSynchronize());
}

__global__ void _k_CACU_BN_GAMMA_GRAD_CUDA(const float_t *_x,
		const float_t *d_y, int num, int length, int channel,
		float_t *d_gamma) {

	int tid = threadIdx.x;
	int bid = blockIdx.x;

	extern __shared__ float_t shared_data[];

	int data_row, data_col;

	int cin_length = length / channel;

	int set;

	for (int i = bid; i < channel; i += BLOCKNUM)
	{
		shared_data[tid] = 0;

		for (int j = tid; j < cin_length * num; j += THREADNUM)
		{
			data_row = j / cin_length;
			data_col = j % cin_length;
			set = data_row * length + data_col + i * cin_length;
			shared_data[tid] += (_x[set] * d_y[set]);
		}

		__syncthreads();

		int acc_length = THREADNUM / 2;
		while (acc_length > 0) {
			if (tid < acc_length)
				shared_data[tid] += shared_data[tid + acc_length];
			acc_length /= 2;
			__syncthreads();
		}

		if (tid == 0)
			d_gamma[i] += shared_data[0];
	}
}

/**
 * @cacu_bn_gamma_grad_cuda
 * calculate the gradient of bn layer's scale
 * _x: is ^x
 * d_y: gradient propagate form top layer
 * length: size of a feature map
 * d_gamma: gradient of gamma
 */
extern "C" void cacu_bn_gamma_grad_cuda(const float_t *_x, const float_t *d_y,
		int num, int length, int channel, float_t *d_gamma) {
	_k_CACU_BN_GAMMA_GRAD_CUDA<<<BLOCKNUM, THREADNUM,
	THREADNUM * sizeof(float_t)>>>(_x, d_y, num, length, channel, d_gamma);
	CUDA_CHECK(hipDeviceSynchronize());
}

__global__ void _k_CACU_SSX_CUDA(const float_t *x, int length, float_t *y) {

	int tid = threadIdx.x;
	int bid = blockIdx.x;

	int threadid = bid * THREADNUM + tid;

	for (int i = threadid; i < length; i += BLOCKNUM * THREADNUM) {
		y[i] *= x[i];
	}
}

/**
 * @cacu_ssx_cuda
 * math y[i] *= x[i] :
 * scale by element wise.
 */
extern "C" void cacu_ssx_cuda(const float_t *x, int length, float_t *y) {
	_k_CACU_SSX_CUDA<<<BLOCKNUM, THREADNUM, 0>>>(x, length, y);
	CUDA_CHECK(hipDeviceSynchronize());
}

__global__ void _k_CACU_GROUP_ALLOC_CUDA(int num, int channel,
		int channel_length, int group, float_t *y) {

	int tid = threadIdx.x;
	int bid = blockIdx.x;

	int length = channel * channel_length;

	int start_set = (channel / group) * channel_length;

	int copy_length = length - start_set;

	float_t *yp, *xp;

	for (int n = bid; n < num; n += BLOCKNUM) {

		yp = y + n * length + start_set;
		xp = y + n * length;

		for (int i = tid; i < copy_length; i += THREADNUM) {
			yp[i] = xp[i % start_set];
		}
	}
}

/**
 * @cacu_group_alloc
 * alloc data by group
 */
extern "C" void cacu_group_alloc_cuda(int num, int channel, int channel_length,
		int group, float_t *y) {

	_k_CACU_GROUP_ALLOC_CUDA<<<BLOCKNUM, THREADNUM, 0>>>(num, channel,
			channel_length, group, y);
	CUDA_CHECK(hipDeviceSynchronize());

}

__global__ void _k_CACU_GROUP_COMBINE_CUDA(int num, int channel,
		int channel_length, int group, float_t *y) {

	int tid = threadIdx.x;
	int bid = blockIdx.x;

	int length = channel * channel_length;

	int start_set = (channel / group) * channel_length;

	float_t *yp, *xp;

	for (int n = bid; n < num; n += BLOCKNUM) {

		yp = y + n * length + start_set;
		xp = y + n * length;

		for (int i = tid; i < start_set; i += THREADNUM) {
			for (int g = 0; g < group - 1; ++g)
				xp[i] += yp[i + g * start_set];
		}
	}
}

/**
 * @cacu_group_combine
 * combine data by group
 */
extern "C" void cacu_group_combine_cuda(int num, int channel,
		int channel_length, int group, float_t *y) {
	_k_CACU_GROUP_COMBINE_CUDA<<<BLOCKNUM, THREADNUM, 0>>>(num, channel,
			channel_length, group, y);
	CUDA_CHECK(hipDeviceSynchronize());

}

}
#endif
#endif
